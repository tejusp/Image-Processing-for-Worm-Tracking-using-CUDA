#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <device_functions.hpp>
#include <>
#include <hip/driver_types.h>
#include <host_defines.h>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda_devptrs.hpp>
#include <opencv2/core/gpumat.hpp>
#include <opencv2/gpu/device/common.hpp>
#include <stdio.h>
#include <hip/hip_vector_types.h>
#include <ostream>
#include <string>
#include <iostream>
#include <typeinfo>
using namespace std;
using std::cout;

__shared__ int edgesValues[240];

__global__ void funcKernel(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
//	printf("test=%d", edgesValues[0]);

	if (rowInd >= rows || colInd >= cols)
		return;
	const float* rowsrcptr = (const float *) (((char *) srcptr)
			+ rowInd * srcstep);
//	float* rowdstPtr = (float *) (((char *) dstptr) + rowInd * dststep);
	float val = rowsrcptr[colInd];
//	printf("test");
//	printf("\nat row=%d col=%d inp array=%d ", rowInd, colInd,
//			inputArray_d[rowInd * cols + colInd]);

	if ((rowInd > 2 && rowInd < (rows - 2))
			&& (colInd > 2 && colInd < (cols - 2))) {
		if (val == 255) {
			const float* rowsrcptrNxt = (const float *) (((char *) srcptr)
					+ (rowInd + 1) * srcstep);
			const float* rowsrcptrPrev = (const float *) (((char *) srcptr)
					+ (rowInd - 1) * srcstep);
			if (rowsrcptrPrev[colInd - 1] == 0 || rowsrcptrPrev[colInd] == 0
					|| rowsrcptrPrev[colInd + 1] == 0
					|| rowsrcptr[colInd - 1] == 0 || rowsrcptr[colInd - 1] == 0
					|| rowsrcptrNxt[colInd - 1] == 0
					|| rowsrcptrNxt[colInd] == 0
					|| rowsrcptrNxt[colInd + 1] == 0) {
				//outputArray_d[rowInd * cols + colInd] = 1;
				edgesValues[rowInd * cols + colInd] = 1;
//				printf("\nat row=%d col=%d out araay=%d ", rowInd, colInd,
//						outputArray_d[rowInd * cols + colInd]);

//				printf(
//						"\nat row=%d col=%d ;val=%f, rowsrcptr[colInd-1]=%f, rowsrcptr[colInd+1]=%f,rowsrcptrNxt =%f",
//						rowInd, colInd, val, rowsrcptr[colInd - 1],
//						rowsrcptr[colInd + 1], rowsrcptrNxt[colInd]);

			} else {
				edgesValues[rowInd * cols + colInd] = 0;
//
//				outputArray_d[rowInd * cols + colInd] = inputArray_d[rowInd
//						* cols + colInd];
			}

		}
	}

	for (int i = 0; i < rows * cols; i++) {
//		printf("in loop=%d", i);
	}

}

__global__ void funcKernel2(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows, int* inputArray_d,
		int* outputArray_d) {

	int rowInd = blockIdx.y * blockDim.y + threadIdx.y;
	int colInd = blockIdx.x * blockDim.x + threadIdx.x;
	if (rowInd >= rows || colInd >= cols)
		return;
//	const float* rowsrcptr = (const float *) (((char *) srcptr)
//			+ rowInd * srcstep);
//	float* rowdstPtr = (float *) (((char *) dstptr) + rowInd * dststep);
//	printf("\nat row=%d col=%d inp array=%d ", rowInd, colInd,
//			inputArray_d[rowInd * cols + colInd]);
//	__shared__ int test[240];

	for (int i = 0; i < 239; i++) {
		if (edgesValues[i] == 1) {
			printf("test contour at %d is %d \n", i, edgesValues[i]);
		}
	}

}

int divUp(int a, int b) {
	return (a + b - 1) / b;
}

//extern "C"
//{
void func(const float* srcptr, float* dstptr, size_t srcstep,
		const size_t dststep, int cols, int rows) {
	dim3 blDim(32, 8);
	dim3 grDim(divUp(cols, blDim.x), divUp(rows, blDim.y));
//	size_t size = sizeof(int);

	int inputArray_h[rows * cols];
	int outputArray_h[rows * cols];
	int* inputArray_d;
	int* outputArray_d;

	for (int j = 0; j < rows * cols; j++) {
		inputArray_h[j] = 0;
	}

//	for (int i = rows * cols - 1; i >= 0; i--)
//		cout << "==" << inputArray_h[i];

	int ARRAY_BYTES = rows * cols * sizeof(int);
//allocate GPU memory

	hipMalloc((void**) &inputArray_d, ARRAY_BYTES);
	hipMalloc((void**) &outputArray_d, ARRAY_BYTES);

//	hipMalloc((void**) &inputMatrix_d, ARRAY_BYTES);
//	hipMalloc((void**) &outputMatrix_d, ARRAY_BYTES);

	hipMemcpy(inputArray_d, inputArray_h, ARRAY_BYTES, hipMemcpyHostToDevice);

	std::cout << "calling kernel from func\n";
	funcKernel<<<grDim, blDim>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			inputArray_d, outputArray_d);
	hipDeviceSynchronize();
	funcKernel2<<<1, 1>>>(srcptr, dstptr, srcstep, dststep, cols, rows,
			inputArray_d, outputArray_d);

	hipMemcpy(outputArray_d, outputArray_h, ARRAY_BYTES,
			hipMemcpyDeviceToHost);

//	if (edgesValues[0]) {
//		cout << "host: " << edgesValues[0] << endl;
//	}

	cout << "\n\nstarting output in host" << endl;

//	for (int i = rows * cols - 1; i >= 0; i--)
//		cout << "==" << (int) outputArray_h[i];

//	int *test;
//	hipMemcpy(counter, test, sizeof(int));
	hipDeviceSynchronize();
//	std::cout << "done with kernel call\n==" << counter << endl;
}
//}
